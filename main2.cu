#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <random>
#include <fstream>
#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>

using namespace std;

// Matrix Vector Multiplication with GPU Streams
#define WARP_SIZE 32
#define FULL_MASK 0xffffffff
#define NWARPS 32
#define ROWS_PER_BLOCK 4

int numStreams = 2;

// randomization
std::random_device rd;
std::mt19937 gen(rd()); // Mersenne Twister 19937 generator
std::uniform_real_distribution<double> distribution(1.0, 100.0);

__global__ void matVecKernel(int m, int n, double *rows, double *vec, double *res) {
    size_t row = threadIdx.x;
    
    if (row < m) {
        // printf("Row: %d\n", (int) row);
        double sum = 0.0;
        int offset = row * n;
        for (int i = 0; i < n; i++) {
            // printf("r,c = (%d, %d)\n", (int) row, i);
            sum += rows[offset + i] * vec[i];
            // printf("%d, %g, %g\n", (row * cols) + i, vector[i], sum);
            // printf("%ld, %g, %g, %g\n", (row * n) + i, rows[offset + i], vec[i], sum);
        }
        res[row] = sum;
    }
}

void instantiateMatVec(int m, int n, double *mat, double *vec) {
  for (int i = 0; i < m; i++) {
    for (int j = 0; j < n; j++) {
        mat[n * i + j] = distribution(gen);  // Generate a random double value and store it in the matrix
    }
  }
  for (int i = 0; i < n; i++) {
    vec[i] = distribution(gen);
  }
}

int main() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    std::cout << "Compute Capability: " << prop.major << "." << prop.minor << std::endl;
    std::cout << "Concurrent Kernels: " << (prop.concurrentKernels ? "Yes" : "No") << std::endl;

    int ncuda_devices = 0;
    hipGetDeviceCount(&ncuda_devices);
    printf("ncuda_devices = %d\n",ncuda_devices);

    if (ncuda_devices == 0) {
        fprintf(stderr,"NO CUDA DEVICES EXITING\n");
        return 0;
    }
    hipSetDevice(0);


    int M = 100;
    int N = 1000;

    int rowsPerBlock = (M / numStreams);

    double mat_h[M*N] = {0};
    double mat_h0[rowsPerBlock*N] = {0};
    double mat_h1[rowsPerBlock*N] = {0};

    double vec_h[N] = {0};
    double res_h0[rowsPerBlock] = {0};
    double res_h1[rowsPerBlock] = {0};

    instantiateMatVec(M, N, mat_h, vec_h);
    instantiateMatVec(rowsPerBlock, N, mat_h0, vec_h);
    instantiateMatVec(rowsPerBlock, N, mat_h1, vec_h);

    // instantiateMatVec(M, N, mat_h, vec_h);

    double *mat_d0;
    double *mat_d1;
    double *vec_d;
    double *res_d0;
    double *res_d1;



    struct timeval startTime;
    struct timeval endTime;

    dim3 nthreads(rowsPerBlock, 1, 1);
    dim3 nblocks (1, 1, 1); 
    
    gettimeofday(&startTime, nullptr);  

    hipStream_t stream0, stream1;
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);

        // allocate memory on device
    hipMalloc( (void**) &mat_d0, sizeof(double)*rowsPerBlock*N);
    hipMalloc( (void**) &mat_d1, sizeof(double)*rowsPerBlock*N);

    hipMalloc( (void**) &vec_d, sizeof(double)*N);
    hipMalloc( (void**) &res_d0, sizeof(double)*rowsPerBlock);
    hipMalloc( (void**) &res_d1, sizeof(double)*rowsPerBlock);

    hipMemcpy(vec_d, vec_h, sizeof(double)*N,hipMemcpyHostToDevice);

    for (int i = 0; i < 10; i++) {
        hipMemcpyAsync(mat_d0, mat_h0, sizeof(double)*rowsPerBlock*N, hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(mat_d1, mat_h1, sizeof(double)*rowsPerBlock*N, hipMemcpyHostToDevice, stream1);
        matVecKernel<<<nblocks, nthreads, 0, stream0>>>(M, N, mat_d0, vec_d, res_d0);
        matVecKernel<<<nblocks, nthreads, 0, stream1>>>(M, N, mat_d1, vec_d, res_d1);

        hipMemcpyAsync(res_h0, res_d0, sizeof(double)*rowsPerBlock, hipMemcpyDeviceToHost, stream0);
        hipMemcpyAsync(res_h1, res_d1, sizeof(double)*rowsPerBlock, hipMemcpyDeviceToHost, stream0);
        // hipMemcpyAsync(&res_h[rowsPerBlock], &res_d[rowsPerBlock], sizeof(double)*min(rowsPerBlock, M - rowsPerBlock), hipMemcpyDeviceToHost, stream1);
    }

    hipStreamSynchronize(stream0);
    hipStreamSynchronize(stream1);

    hipStreamDestroy(stream0);
    hipStreamDestroy(stream1);

    gettimeofday(&endTime, nullptr);

    int microseconds = (endTime.tv_sec - startTime.tv_sec) * 1000000 + (endTime.tv_usec - startTime.tv_usec);
    std::cout << "creating streams took " << microseconds << " microseconds" << std::endl;

    double expected[M] = {0};
    for (int i = 0; i < M; i++) {
        double sum = 0.0;
        int offset = i * N;
        for (int j = 0; j < N; j++) {
            sum += mat_h[offset+j] * vec_h[j];
        }
        expected[i] = sum;
    }

    for (int i = 0; i < rowsPerBlock; i++) {
        if (abs(expected[i] - res_h0[i]) > 0.0001) {
            printf("DIFF FOUND: expected: %g, actual: %g", expected[i], res_h0[i]);
        } else if (i % 100 == 0) {
            printf("no diff, found %g \n", expected[i]);
        }
    }
    for (int i = 0; i < rowsPerBlock; i++) {
        if (abs(expected[i+rowsPerBlock] - res_h1[i]) > 0.0001) {
            printf("DIFF FOUND: expected: %g, actual: %g", expected[i+rowsPerBlock], res_h1[i]);
        } else if (i % 100 == 0) {
            printf("no diff, found %g \n", expected[i+rowsPerBlock]);
        }
    }

    hipFree(res_d0);
    hipFree(res_d1);
    hipFree(vec_d);
    hipFree(mat_d0);
    hipFree(mat_d1);
}