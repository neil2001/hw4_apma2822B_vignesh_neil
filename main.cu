#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <random>
#include <fstream>
#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>

using namespace std;

// Matrix Vector Multiplication with GPU Streams
#define WARP_SIZE 32
#define FULL_MASK 0xffffffff
#define NWARPS 32
#define ROWS_PER_BLOCK 4

int numStreams = 8;

// randomization
std::random_device rd;
std::mt19937 gen(rd()); // Mersenne Twister 19937 generator
std::uniform_real_distribution<double> distribution(1.0, 100.0);

__global__ void matVecKernel(int m, int n, double *rows, double *vec, double *res) {
    size_t row = threadIdx.x;
    
    if (row < m) {
        // printf("Row: %d\n", (int) row);
        double sum = 0.0;
        int offset = row * n;
        for (int i = 0; i < n; i++) {
            // printf("r,c = (%d, %d)\n", (int) row, i);
            sum += rows[offset + i] * vec[i];
            // printf("%d, %g, %g\n", (row * cols) + i, vector[i], sum);
        }
        res[row] = sum;
    }
}

void instantiateMatVec(int m, int n, double *mat, double *vec) {
  for (int i = 0; i < m; i++) {
    for (int j = 0; j < n; j++) {
        mat[n * i + j] = distribution(gen);  // Generate a random double value and store it in the matrix
    }
  }
  for (int i = 0; i < n; i++) {
    vec[i] = distribution(gen);
  }
}

int main() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    std::cout << "Compute Capability: " << prop.major << "." << prop.minor << std::endl;
    std::cout << "Concurrent Kernels: " << (prop.concurrentKernels ? "Yes" : "No") << std::endl;

    int ncuda_devices = 0;
    hipGetDeviceCount(&ncuda_devices);
    printf("ncuda_devices = %d\n",ncuda_devices);

    if (ncuda_devices == 0) {
        fprintf(stderr,"NO CUDA DEVICES EXITING\n");
        return 0;
    }
    hipSetDevice(0);


    int M = 1000;
    int N = 10000;

    double mat_h[M*N] = {0};
    double vec_h[N] = {0};
    double res_h[M] = {0};

    instantiateMatVec(M, N, mat_h, vec_h);

    double *mat_d;
    double *vec_d;
    double *res_d;

    // allocate memory on device
    hipMalloc( (void**) &mat_d, sizeof(double)*M*N);
    hipMalloc( (void**) &vec_d, sizeof(double)*N);
    hipMalloc( (void**) &res_d, sizeof(double)*M);

    hipMemcpy(vec_d, vec_h, sizeof(double)*N,hipMemcpyHostToDevice);

    struct timeval startTime;
    struct timeval endTime;


    hipStream_t streams[numStreams];

    int rowsPerBlock = (M / numStreams) + 1;

    for (int i=0; i<numStreams; i++) {
        hipStreamCreate(&streams[i]);
    }

    dim3 nthreads(rowsPerBlock, 1, 1); // threads per block NOTE NOT MORE THAN 1024
    // 30 rows per block
    // 1 thread per row

    dim3 nblocks (1, 1, 1); // blocks per grid -> should be 1
    // dim3 nblocks ((rowsPerBlock + nthreads.x-1)/nthreads.x, 1, 1); // blocks per grid -> should be 1
    gettimeofday(&startTime, nullptr);  

    for (int i=0; i<numStreams; i++) {
        // copy H2D
        int numToCpy = min(N*M - i*rowsPerBlock*N, rowsPerBlock*N);
        hipMemcpyAsync(&mat_d[i*rowsPerBlock*N], &mat_h[i*rowsPerBlock*N], sizeof(double)*numToCpy, hipMemcpyHostToDevice, streams[i]);
        // run / kernel func
        matVecKernel<<<nblocks, nthreads, 0, streams[i]>>>(M, N, &mat_d[i*rowsPerBlock*N], vec_d, &res_d[i*rowsPerBlock]);
        // hipMemcpyAsync(mat_h[], mat_d[], sizeof(double)*, hipMemcpyDeviceToDevice, streams[i]);
        hipMemcpyAsync(&res_h[i*rowsPerBlock], &res_d[i*rowsPerBlock], sizeof(double)*min(rowsPerBlock, M - i * rowsPerBlock), hipMemcpyDeviceToHost, streams[i]);
        // copy D2H
    }
    
    for (int i = 0; i < numStreams; i++) {
        hipStreamSynchronize(streams[i]);
    }
    for (int i = 0; i < numStreams; i++) {
        hipStreamDestroy(streams[i]);
    }

    hipDeviceSynchronize();

    gettimeofday(&endTime, nullptr);

    int microseconds = (endTime.tv_sec - startTime.tv_sec) * 1000000 + (endTime.tv_usec - startTime.tv_usec);
    std::cout << "creating streams took " << microseconds << " microseconds" << std::endl;

    double expected[M] = {0};
    for (int i = 0; i < M; i++) {
        double sum = 0.0;
        int offset = i * N;
        for (int j = 0; j < N; j++) {
            sum += mat_h[offset+j] * vec_h[j];
        }
        expected[i] = sum;
    }

    for (int i = 0; i < M; i++) {
        if (abs(expected[i] - res_h[i]) > 0.0001) {
            printf("DIFF FOUND: expected: %g, actual: %g", expected[i], res_h[i]);
        } else if (i % 100 == 0) {
            printf("no diff, found %g \n", expected[i]);
        }
    }
    hipFree(res_d);
    hipFree(vec_d);
    hipFree(mat_d);
}